#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void w(int *data, const int val, const int sz){
  for (int i = threadIdx.x+blockDim.x*blockIdx.x; i< sz; i+=gridDim.x*blockDim.x)
    data[i] = val;
}

__global__ void r(int *data, int *r, const int sz){
  int val;
  for (int i = threadIdx.x+blockDim.x*blockIdx.x; i< sz; i+=gridDim.x*blockDim.x)
    val += data[i];
  if (val == 0) *r = val;
}

int main(){
  const int s = 1024*1024;  // 1M
  const int sz = s*sizeof(int);  // 4MB
  int *d1, *d2, *res;
  hipMalloc(&d1, sz*10);
  hipMalloc(&d2, sz*10);
  hipMalloc(&res, sizeof(int));
  hipMemset(d1, 1, sz);
  hipMemset(d2, 1, sz);
  w<<<160,1024>>>(d2, 1, s);
  r<<<160,1024>>>(d1, res, s);
  w<<<160,1024>>>(d1, 1, s);
  r<<<160,1024>>>(d1, res, s);
  hipDeviceSynchronize();
}
